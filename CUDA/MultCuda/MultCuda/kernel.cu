
#include "hip/hip_runtime.h"


#include <stdio.h>

#include <cstdlib>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>

using namespace std;

template <typename T> struct mat {
	int rows;
	int cols;
	vector<T> data;
};

hipError_t multMatrices(mat<float> *c, mat<float> a, mat<float> b, unsigned int threads, unsigned int blocks);

__global__ void multKernel(float *matr, const float *mat1, const float *mat2, const int width)
{
    int tId = threadIdx.x;
	int bId = blockIdx.x;
	int cols = blockDim.x;
    //c[i] = a[i] + b[i];
	//int col = get_global_id(0);
	//int row = get_global_id(1);

	float result = 0;

	for (int i = 0; i < width; ++i){
		float val1 = mat1[bId*width + i];
		float val2 = mat2[i * cols + tId];
		result += val1 * val2;
		//printf("%i - %i - %f, %f, %f\n", bId*width + i, i * cols + tId, val1, val2, result);
	}
	//printf("%i, %i, %f\n", bId, tId, result);
	matr[bId * cols + tId] = result;
}

template <typename T> void print_mat(mat<T> &mt){
	for (int i = 0, l = mt.rows; i<l; ++i){
		for (int j = 0, m = mt.cols; j<m; ++j){
			cout << mt.data[i*m + j] << " ";
		}
		cout << endl;
	}
}

template <typename T> bool read_mat(string filename, mat<T> &mt){
	ifstream file;

	file.open(filename);
	if (!file.is_open())
		return false;

	int rows, cols;
	file >> rows >> cols;
	mt.rows = rows;
	mt.cols = cols;

	for (int i = 0; i<rows; ++i){
		for (int j = 0; j<cols; ++j){
			T value;
			file >> value;
			mt.data.push_back(value);
			if (file.eof())
				return false;
		}
	}

	return true;
}

template <typename T> void init_mat(int rows, int cols, mat<T> &mt){
	mt.rows = rows;
	mt.cols = cols;
	mt.data.assign(rows*cols, 0);
}

int main(int argc, char **argv) {
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };
	mat<float> mat1, mat2, matr;

	if (argc < 3) {
		return (cerr << "Usage: multmat mat1 mat2 [result only:(1|0) supress:2]" << endl), 1;
	}

	auto start = std::chrono::system_clock::now();

	bool notresultonly = true;
	bool noresult = false;

	if (argc > 3){
		notresultonly = (argv[3][0] == '0');
		noresult = (argv[3][0] == '2');
	}

	read_mat(argv[1], mat1);
	read_mat(argv[2], mat2);
	init_mat(mat1.rows, mat2.cols, matr);

    // Add vectors in parallel.
    hipError_t cudaStatus = multMatrices(&matr, mat1, mat2, mat2.cols, mat1.rows);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	if (!noresult){
		if (notresultonly){
			cout << "Mat 1" << endl;
			print_mat(mat1);
			cout << endl << "Mat 2" << endl;
			print_mat(mat2);
			cout << endl << "Result" << endl;
		}
		else
			cout << matr.rows << " " << matr.cols << endl;
		print_mat(matr);
	}

	auto end = std::chrono::system_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
	cerr << endl << "time: " << elapsed.count() << "ms" << endl;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multMatrices(mat<float> *c, mat<float> a, mat<float> b, unsigned int threads, unsigned int blocks)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
	float size = threads * blocks;
    hipError_t cudaStatus;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_a, a.cols * a.rows * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_b, b.cols * b.rows * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a.data.data(), a.cols * a.rows * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_b, b.data.data(), b.cols * b.rows * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    multKernel<<< threads, blocks >>>(dev_c, dev_a, dev_b, a.cols);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c->data.data(), dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
